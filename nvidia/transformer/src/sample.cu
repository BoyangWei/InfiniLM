#include "hip/hip_runtime.h"
﻿#include <cub/device/device_reduce.cuh>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh>
#include <hip/hip_fp16.h>

extern "C" hipError_t argmax_half(
    void *temp_storage, size_t *temp_storage_bytes,
    half const *input, int num_items,
    hipcub::KeyValuePair<int, half> *output,
    hipStream_t stream)
{
    return hipcub::DeviceReduce::ArgMax(
        temp_storage, *temp_storage_bytes,
        input,
        output,
        num_items,
        stream);
}

extern "C" hipError_t radix_sort_half(
    void *temp_storage, size_t *temp_storage_bytes,
    half const *key_in, half *key_out,
    unsigned int const *value_in, unsigned int *value_out,
    int num_items,
    hipStream_t stream)
{
    return hipcub::DeviceRadixSort::SortPairsDescending(
        temp_storage, *temp_storage_bytes,
        key_in,
        key_out,
        value_in,
        value_out,
        num_items,
        0,
        sizeof(half) * 8,
        stream);
}

extern "C" hipError_t inclusive_sum_half(
    void *temp_storage, size_t *temp_storage_bytes,
    half *data, int num_items,
    hipStream_t stream)
{
    return hipcub::DeviceScan::InclusiveSum(
        temp_storage, *temp_storage_bytes,
        data,
        data,
        num_items,
        stream);
}

#define RUNTIME(statement)                                                                      \
    {                                                                                           \
        auto error = statement;                                                                 \
        if (error != hipSuccess)                                                               \
        {                                                                                       \
            printf("Error: %s (%d) at \"%s\"\n", hipGetErrorString(error), error, #statement); \
            return error;                                                                       \
        }                                                                                       \
    }

static __global__ void partial_softmax_half_kernel(
    half *__restrict__ data,
    half const *__restrict__ max_,
    float temperature,
    int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        data[i] = hexp((data[i] - __ldg(max_)) / (half)temperature);
    }
}

extern "C" hipError_t partial_softmax_half(
    half *data,
    float temperature,
    int topk,
    hipStream_t stream)
{
    half *max_ = nullptr;
    RUNTIME(hipMallocAsync(&max_, sizeof(half), stream))
    RUNTIME(hipMemcpyAsync(max_, data, sizeof(half), hipMemcpyDeviceToDevice, stream))

    auto block = min(1024, topk);
    auto grid = (topk + block - 1) / block;
    partial_softmax_half_kernel<<<grid, block, 0, stream>>>(data, max_, temperature, topk);

    RUNTIME(hipFreeAsync(max_, stream))
    return hipGetLastError();
}

static __global__ void random_sample_kernel(
    half const *__restrict__ data,
    unsigned int const *__restrict__ indices,
    unsigned int *__restrict__ index_,
    float probability,
    int n)
{
    half p = probability * (float)data[n - 1];
    for (int i = 0; i < n; ++i)
    {
        if (data[i] >= p)
        {
            *index_ = indices[i];
            return;
        }
    }
}

extern "C" hipError_t random_sample_half(
    half const *data,
    unsigned int const *indices,
    unsigned int *index,
    float probability,
    int topk,
    hipStream_t stream)
{
    unsigned int *index_ = nullptr;
    hipMallocAsync(&index_, sizeof(unsigned int), stream);

    random_sample_kernel<<<1, 1, 0, stream>>>(data, indices, index_, probability, topk);

    hipMemcpy(index, index_, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipFree(index_);

    return hipGetLastError();
}
